#include "hip/hip_runtime.h"
/*
 * batchnorm.cu
 *
 *  Created on: Jan 5, 2023
 *      Author: Maciej Kozarzewski
 */

#include <minml/backend/cuda_backend.h>
#include <minml/backend/backend_utils.hpp>

#include "../utils.hpp"

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <cmath>
#include <algorithm>
#include <cassert>

namespace
{
	__device__ float get_mean(const float *ptr, int idx, int last_dim)
	{
		assert(idx >= 0 && idx < last_dim);
		return ptr[idx];
	}
	__device__ float get_stddev(const float *ptr, int idx, int last_dim)
	{
		assert(idx >= 0 && idx < last_dim);
		return std::sqrt(ptr[last_dim + idx] + 1.0e-6f);
	}
	__device__ float get_gamma(const float *ptr, int idx, int last_dim)
	{
		assert(idx >= 0 && idx < last_dim);
		return ptr[2 * last_dim + idx];
	}
	__device__ float get_beta(const float *ptr, int idx, int last_dim)
	{
		assert(idx >= 0 && idx < last_dim);
		return ptr[3 * last_dim + idx];
	}

	template<typename T>
	__device__ T square(T x)
	{
		return x * x;
	}

	/*
	 * Welford's online algorithm for calculating mean and variance
	 */
	template<typename T>
	class AvgVarStats
	{
			T samples = static_cast<T>(0);
			T M = static_cast<T>(0); // mean
			T M2 = static_cast<T>(0); // variance
		public:
			__device__ void add(T x) noexcept
			{
				samples += static_cast<T>(1);
				const T delta = x - M;
				M += delta / samples;
				M2 += delta * (x - M);
			}
			__device__ T get_average() const noexcept
			{
				return M;
			}
			__device__ T get_variance() const noexcept
			{
				assert(samples >= static_cast<T>(2));
				return M2 / (samples - static_cast<T>(1));
			}

			__device__ void merge_with(const AvgVarStats<T> &rhs) noexcept
			{
				assert(this->samples >= static_cast<T>(0) && rhs.samples >= static_cast<T>(0));
				if (rhs.samples == static_cast<T>(0))
					return;
				if (this->samples == static_cast<T>(0))
				{
					this->samples = rhs.samples;
					this->M = rhs.M;
					this->M2 = rhs.M2;
				}
				else
				{
					const T total_samples = this->samples + rhs.samples;
					const T total_M = (this->samples * this->M + rhs.samples * rhs.M) / total_samples;
					const T total_M2 = this->M2 + rhs.M2 + square(this->M - rhs.M) * (this->samples * rhs.samples) / total_samples;
					this->samples = total_samples;
					this->M = total_M;
					this->M2 = total_M2;
				}
			}
	};

	using namespace ml;
	__device__ void combine_stats(AvgVarStats<float> *stats)
	{
		assert(blockDim.x == 32 && blockDim.y == 32);
		for (int i = 16; i >= 1; i /= 2)
		{
			if (threadIdx.y < i)
				stats[threadIdx.y * 32 + threadIdx.x].merge_with(stats[(i + threadIdx.y) * 32 + threadIdx.x]);
			__syncthreads();
		}
	}
	__global__ void kernel_batchnorm_forward_avg_var_1(AvgVarStats<float> *__restrict__ workspace, const float *__restrict__ input, int first_dim,
			int last_dim)
	{
		assert(blockDim.x == 32 && blockDim.y == 32);
		__shared__ AvgVarStats<float> shared_stats[32 * 32]; // 32 x 3 layout will be perfectly interleaved with no bank conflicts

		const int tid = blockIdx.x * 32 + threadIdx.x;

		AvgVarStats<float> thread_stat;
		if (tid < last_dim)
			for (int i = 32 * blockIdx.y + threadIdx.y; i < first_dim; i += 32 * gridDim.y)
				thread_stat.add(input[i * last_dim + tid]);

		shared_stats[threadIdx.y * 32 + threadIdx.x] = thread_stat;
		__syncthreads();

		combine_stats(shared_stats);
		if (threadIdx.y == 0 and tid < last_dim)
			workspace[blockIdx.y * last_dim + tid] = shared_stats[threadIdx.x];
	}
	__global__ void kernel_batchnorm_forward_avg_var_2(float *__restrict__ running_stat, const AvgVarStats<float> *__restrict__ workspace,
			int first_dim, int last_dim)
	{
		assert(blockDim.x == 32 && blockDim.y == 32);
		__shared__ AvgVarStats<float> shared_stats[32 * 32]; // 32 x 3 layout will be perfectly interleaved with no bank conflicts

		const int tid = blockIdx.x * 32 + threadIdx.x;

		AvgVarStats<float> thread_stat;
		if (tid < last_dim)
			for (int i = threadIdx.y; i < first_dim; i += 32)
				thread_stat.merge_with(workspace[i * last_dim + tid]);

		shared_stats[threadIdx.y * 32 + threadIdx.x] = thread_stat;
		__syncthreads();

		combine_stats(shared_stats);
		if (threadIdx.y == 0 and tid < last_dim)
		{
			running_stat[tid] = shared_stats[threadIdx.x].get_average();
			running_stat[last_dim + tid] = shared_stats[threadIdx.x].get_variance();
		}
	}

	__global__ void kernel_batchnorm_forward(const float *weights, const float *input, float *output, const float *running_stats, int2 shape,
			mlActivationType_t act)
	{
		const int tid = blockIdx.x * blockDim.x + threadIdx.x;
		if (tid < shape.y)
		{
			/* weights rows are:
			 * mean
			 * variance
			 * gamma
			 * beta
			 */
			const float mean = get_mean(running_stats, tid, shape.y);
			const float stddev = get_stddev(running_stats, tid, shape.y);
			const float gamma = get_gamma(weights, tid, shape.y);
			const float beta = get_beta(weights, tid, shape.y);

			const float scale = gamma / stddev;
			const float shift = -mean * scale + beta;

			for (int i = blockIdx.y * blockDim.y + threadIdx.y; i < shape.x; i += gridDim.y * blockDim.y)
			{
				float tmp = input[i * shape.y + tid] * scale + shift;
				if (act == ACTIVATION_RELU)
					tmp = max(0.0f, tmp);
				if (act == ACTIVATION_TANH)
					tmp = tanh(tmp);
				if (act == ACTIVATION_SIGMOID)
					tmp = 1.0f / (1.0f + exp(-tmp));
				output[i * shape.y + tid] = tmp;
			}
		}
	}

	__global__ void kernel_batchnorm_inference(const float *weights, const float *input, float *output, int2 shape, mlActivationType_t act)
	{
		const int tid = blockIdx.x * blockDim.x + threadIdx.x;
		if (tid < shape.y)
		{
			const float mean = get_mean(weights, tid, shape.y);
			const float stddev = get_stddev(weights, tid, shape.y);
			const float gamma = get_gamma(weights, tid, shape.y);
			const float beta = get_beta(weights, tid, shape.y);

			const float scale = gamma / stddev;
			const float shift = -mean * scale + beta;

			for (int i = blockIdx.y * blockDim.y + threadIdx.y; i < shape.x; i += gridDim.y * blockDim.y)
			{
				float tmp = input[i * shape.y + tid] * scale + shift;
				if (act == ACTIVATION_RELU)
					tmp = max(0.0f, tmp);
				if (act == ACTIVATION_TANH)
					tmp = tanh(tmp);
				if (act == ACTIVATION_SIGMOID)
					tmp = 1.0f / (1.0f + exp(-tmp));
				output[i * shape.y + tid] = tmp;
			}
		}
	}

	__device__ void reduce_add_32x32_dual(float *ptr1, float *ptr2)
	{
		assert(blockDim.x == 32 && blockDim.y == 32);
		for (int i = 16; i >= 1; i /= 2) // sum results stored in temporary array
		{
			if (threadIdx.y < i)
			{
				ptr1[threadIdx.y * 32 + threadIdx.x] += ptr1[(i + threadIdx.y) * 32 + threadIdx.x];
				ptr2[threadIdx.y * 32 + threadIdx.x] += ptr2[(i + threadIdx.y) * 32 + threadIdx.x];
			}
			__syncthreads();
		}
	}
	__global__ void kernel_batchnorm_backward_delta_1(float *workspace, const float *input, const float *output, float *gradient_next,
			const float *running_stats, int2 shape, mlActivationType_t act)
	{
		__shared__ float d_sigma[32 * 32];
		__shared__ float d_mu[32 * 32];
		const int tid = blockIdx.x * 32 + threadIdx.x;

		float d_sigma_acc = 0.0f, d_mu_acc = 0.0f;
		if (tid < shape.y)
		{
			const float mean = get_mean(running_stats, tid, shape.y);
			const float stddev = get_stddev(running_stats, tid, shape.y);
			for (int i = 32 * blockIdx.y + threadIdx.y; i < shape.x; i += 32 * gridDim.y)
			{
				const int tmp_idx = i * shape.y + tid;
				if (act == ACTIVATION_RELU and output[tmp_idx] == 0.0f)
					gradient_next[tmp_idx] = 0.0f;
				if (act == ACTIVATION_TANH)
					gradient_next[tmp_idx] *= (1.0f - output[tmp_idx]) * (1.0f + output[tmp_idx]);
				if (act == ACTIVATION_SIGMOID)
					gradient_next[tmp_idx] *= output[tmp_idx] * (1.0f - output[tmp_idx]);
				d_sigma_acc += gradient_next[tmp_idx] * (input[tmp_idx] - mean) / stddev;
				d_mu_acc += gradient_next[tmp_idx];
			}
		}
		d_sigma[threadIdx.y * 32 + threadIdx.x] = d_sigma_acc;
		d_mu[threadIdx.y * 32 + threadIdx.x] = d_mu_acc;

		__syncthreads();
		reduce_add_32x32_dual(d_sigma, d_mu);
		if (threadIdx.y == 0 and tid < shape.y)
		{
			workspace[2 * blockIdx.y * shape.y + tid] = d_sigma[threadIdx.x];
			workspace[(2 * blockIdx.y + 1) * shape.y + tid] = d_mu[threadIdx.x];
		}
	}
	__global__ void kernel_batchnorm_backward_delta_2(float *workspace, int2 shape)
	{
		__shared__ float storage_d_sigma[32 * 32];
		__shared__ float storage_d_mu[32 * 32];
		const int tid = blockIdx.x * 32 + threadIdx.x;
		float d_sigma = 0.0f, d_mu = 0.0f;
		if (tid < shape.y)
			for (int i = 32 * blockIdx.y + threadIdx.y; i < shape.x; i += 32 * gridDim.y)
			{
				d_sigma += workspace[i * 2 * shape.y + tid];
				d_mu += workspace[(i * 2 + 1) * shape.y + tid];
			}
		storage_d_sigma[threadIdx.y * 32 + threadIdx.x] = d_sigma;
		storage_d_mu[threadIdx.y * 32 + threadIdx.x] = d_mu;

		__syncthreads();
		reduce_add_32x32_dual(storage_d_sigma, storage_d_mu);
		if (threadIdx.y == 0 and tid < shape.y)
		{
			workspace[tid] = storage_d_sigma[threadIdx.x];
			workspace[shape.y + tid] = storage_d_mu[threadIdx.x];
		}
	}

	__global__ void kernel_batchnorm_backward_1(const float *workspace, const float *input, float *gradient_prev, const float *gradient_next,
			const float *weights, float *weight_update, const float *running_stats, int2 shape)
	{
		// avg, stddev, gamma, d_sigma, d_mu
		const int tid = blockIdx.x * blockDim.x + threadIdx.x;
		if (tid < shape.y)
		{
			const float mean = get_mean(running_stats, tid, shape.y);
			const float stddev = get_stddev(running_stats, tid, shape.y);
			const float gamma = get_gamma(weights, tid, shape.y);

			float d_sigma = workspace[tid];
			float d_mu = workspace[shape.y + tid];
			if (blockIdx.y == 0 and threadIdx.y == 0)
			{ // only single line can update this
				weight_update[2 * shape.y + tid] += d_sigma; // gamma
				weight_update[3 * shape.y + tid] += d_mu; // beta
			}

			d_sigma = -gamma / stddev * d_sigma / static_cast<float>(shape.x);
			d_mu = -gamma / stddev * d_mu / static_cast<float>(shape.x);
			for (int i = blockIdx.y * blockDim.y + threadIdx.y; i < shape.x; i += gridDim.y * blockDim.y)
				gradient_prev[i * shape.y + tid] = gamma / stddev * gradient_next[i * shape.y + tid]
						+ d_sigma * (input[i * shape.y + tid] - mean) / stddev + d_mu;
		}
	}

	__global__ void kernel_batchnorm_update(const float *running_stat, float *weights, int first_dim, int last_dim, bool use_gamma, bool use_beta)
	{
		const int tid = blockIdx.x * blockDim.x + threadIdx.x;
		if (tid < last_dim)
		{
			float mean_avg = 0.0f, mean_var = 0.0f;
			for (int i = 0; i < first_dim; i++)
			{
				mean_avg += running_stat[i * 2 * last_dim + tid];
				mean_var += running_stat[(i * 2 + 1) * last_dim + tid];
			}
			weights[0 * last_dim + tid] = mean_avg / first_dim; // running mean average
			weights[1 * last_dim + tid] = mean_var / first_dim; // running mean variance
			if (not use_gamma)
				weights[2 * last_dim + tid] = 1.0f; // gamma
			if (not use_beta)
				weights[3 * last_dim + tid] = 0.0f; // beta
		}
	}

	__global__ void kernel_fold_batchnorm(int first_dim, int last_dim, float *layer_weights, float *layer_bias, const float *batchnorm_weights)
	{
		const float mean = get_mean(batchnorm_weights, blockIdx.x, first_dim);
		const float stddev = get_stddev(batchnorm_weights, blockIdx.x, first_dim);
		const float gamma = get_gamma(batchnorm_weights, blockIdx.x, first_dim);
		const float beta = get_beta(batchnorm_weights, blockIdx.x, first_dim);

		const float scale = gamma / stddev;
		const float shift = -mean * scale + beta;
		for (int i = threadIdx.x; i < last_dim; i += blockDim.x)
			layer_weights[blockIdx.x * last_dim + i] *= scale;

		if (threadIdx.x == 0)
			layer_bias[blockIdx.x] = layer_bias[blockIdx.x] * scale + shift;
	}
}

namespace ml
{
	void cuda_batchnorm_inference(mlContext_t context, mlShape_t shape, const void *input, void *output, const void *weights, mlActivationType_t act)
	{
		const int first_dim = volume_without_last_dim(shape);
		const int last_dim = get_last_dim(shape);
		int2 dim { first_dim, last_dim };

		dim3 blockDim(32, 8);
		dim3 gridDim((last_dim + 31) / 32, std::min(1024, (first_dim + 7) / 8));
		kernel_batchnorm_inference<<<gridDim, blockDim, 0, cuda::Context::getStream(context)>>>(getPointer<float>(weights), getPointer<float>(input),
				getPointer<float>(output), dim, act);
		assert(hipGetLastError() == hipSuccess);
	}
	void cuda_batchnorm_forward(mlContext_t context, mlShape_t shape, const void *input, void *output, void *weights, void *running_stats,
			int running_stat_idx, mlActivationType_t act)
	{
		const int first_dim = volume_without_last_dim(shape);
		const int last_dim = get_last_dim(shape);

		AvgVarStats<float> *workspace = cuda::Context::getWorkspace<AvgVarStats<float>>(context);
		const int workspace_first_dim = std::min((size_t) 256, cuda::Context::getWorkspaceSize(context) / (sizeof(AvgVarStats<float> ) * last_dim));
		assert(workspace_first_dim > 0);

		float *running_stats_ptr = getPointer<float>(running_stats) + running_stat_idx * 2 * last_dim;

		dim3 blockDim(32, 32);
		dim3 gridDim1((last_dim + 31) / 32, workspace_first_dim);

		int2 shape1 { first_dim, last_dim };
		dim3 gridDim2(gridDim1.x);
		int2 shape2 { workspace_first_dim, last_dim };
		hipStream_t stream = cuda::Context::getStream(context);

		kernel_batchnorm_forward_avg_var_1<<<gridDim1, blockDim, 0,stream >>>(workspace, getPointer<float>(input), first_dim, last_dim);
		assert(hipGetLastError() == hipSuccess);
		kernel_batchnorm_forward_avg_var_2<<<gridDim2, blockDim, 0, stream>>>(running_stats_ptr, workspace, workspace_first_dim, last_dim);
		assert(hipGetLastError() == hipSuccess);

		dim3 blockDim3(32, 8);
		dim3 gridDim3((last_dim + 31) / 32, std::min(1024, (first_dim + 7) / 8));
		kernel_batchnorm_forward<<<gridDim3, blockDim3, 0, stream>>>(getPointer<float>(weights), getPointer<float>(input), getPointer<float>(output),
				running_stats_ptr, shape1, act);

		assert(hipGetLastError() == hipSuccess);
	}
	void cuda_batchnorm_backward(mlContext_t context, mlShape_t shape, const void *input, const void *output, void *gradient_prev,
			void *gradient_next, const void *weights, void *weights_update, const void *running_stats, int running_stat_idx, mlActivationType_t act)
	{
		const int first_dim = volume_without_last_dim(shape);
		const int last_dim = get_last_dim(shape);

		float *workspace = cuda::Context::getWorkspace<float>(context);
		const int workspace_first_dim = std::min((size_t) 256, cuda::Context::getWorkspaceSize(context) / (sizeof(float) * last_dim));

		const float *running_stats_ptr = getPointer<float>(running_stats) + running_stat_idx * 2 * last_dim;

		dim3 blockDim(32, 32);
		dim3 gridDim1((last_dim + 31) / 32, workspace_first_dim);

		int2 shape1 { first_dim, last_dim };
		dim3 gridDim2(gridDim1.x);
		int2 shape2 { workspace_first_dim, last_dim };

		hipStream_t stream = cuda::Context::getStream(context);

		kernel_batchnorm_backward_delta_1<<<gridDim1, blockDim, 0, stream>>>(workspace, getPointer<float>(input), getPointer<float>(output),
				getPointer<float>(gradient_next), running_stats_ptr, shape1, act);
		assert(hipGetLastError() == hipSuccess);

		kernel_batchnorm_backward_delta_2<<<gridDim2, blockDim, 0, stream>>>(workspace, shape2);
		assert(hipGetLastError() == hipSuccess);

		dim3 blockDim3(32, 8);
		dim3 gridDim3((last_dim + 31) / 32, std::min(1024, (first_dim + 7) / 8));
		kernel_batchnorm_backward_1<<<gridDim3, blockDim3, 0, stream>>>(workspace, getPointer<float>(input), getPointer<float>(gradient_prev),
				getPointer<float>(gradient_next), getPointer<float>(weights), getPointer<float>(weights_update), running_stats_ptr, shape1);
		assert(hipGetLastError() == hipSuccess);
	}
	void cuda_batchnorm_update(mlContext_t context, mlShape_t shape, const void *running_stat, void *weights, bool use_gamma, bool use_beta)
	{
		const int first_dim = get_first_dim(shape);
		const int last_dim = get_last_dim(shape) / 2;
		int2 dim { first_dim, last_dim };

		dim3 blockDim(256);
		dim3 gridDim(std::max(1u, (last_dim + blockDim.x - 1) / blockDim.x));
		kernel_batchnorm_update<<<gridDim, blockDim, 0, cuda::Context::getStream(context)>>>(getPointer<float>(running_stat),
				getPointer<float>(weights), first_dim, last_dim, use_gamma, use_beta);
		assert(hipGetLastError() == hipSuccess);
	}
	void cuda_fold_batchnorm(mlContext_t context, mlShape_t shape, void *layer_weights, void *layer_bias, const void *batchnorm_weights)
	{
		const int first_dim = get_first_dim(shape);
		const int last_dim = volume_without_first_dim(shape);
		dim3 blockDim(256);
		dim3 gridDim(first_dim);

		kernel_fold_batchnorm<<<gridDim, blockDim, 0, cuda::Context::getStream(context)>>>(first_dim, last_dim, getPointer<float>(layer_weights),
				getPointer<float>(layer_bias), getPointer<float>(batchnorm_weights));
		assert(hipGetLastError() == hipSuccess);
	}

} /* namespace ml */

