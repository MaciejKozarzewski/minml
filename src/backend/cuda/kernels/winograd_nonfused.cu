#include "hip/hip_runtime.h"
/*
 * winograd.cu
 *
 *  Created on: Jan 5, 2023
 *      Author: Maciej Kozarzewski
 */

#include <minml/backend/cuda_backend.h>
#include <minml/backend/backend_utils.hpp>

#include "winograd_transforms.cuh"
#include "../utils.hpp"

#include "../helpers/indexers.cuh"
#include "../helpers/lines_and_tiles.cuh"
#include "../vec/vec_headers.cuh"

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include <cassert>
#include <iostream>
#include <vector>
#include <array>
#include <cmath>
#include <iostream>

namespace
{
	using namespace ml;
	using namespace vectors;

	__device__ float to_fp32(const vec1f x)
	{
		return x.x0;
	}
	__device__ float to_fp32(const vec1h x)
	{
		return static_cast<float>(x.x0);
	}

	__device__ bool is_inside(int h, int w, int height, int width)
	{
		return 0 <= h && h < height && 0 <= w && w < width;
	}

	template<int KernelSize, int TransformSize, typename T>
	__global__ void kernel_transform_weights(T *__restrict__ matrices, const T *__restrict__ weights, int output_filters, int input_filters,
			bool invert)
	{
		constexpr int TileSize = KernelSize + TransformSize - 1;

		Tile<T, KernelSize, KernelSize> tile;
		for (int f = threadIdx.x; f < input_filters; f += blockDim.x)
		{
			const Indexer<4> weights_indexer(output_filters, KernelSize, KernelSize, input_filters);
			for (int col = 0; col < tile.columns(); col++)
				for (int row = 0; row < tile.rows(); row++)
				{
					T tmp;
					if (invert)
						tmp = weights[weights_indexer.at(blockIdx.y, KernelSize - 1 - row, KernelSize - 1 - col, f)];
					else
						tmp = weights[weights_indexer.at(blockIdx.y, row, col, f)];
					tile.at(col, row) = tmp;
				}

			const Indexer<4> matrices_indexer(TileSize, TileSize, output_filters, input_filters);

			const Transform<TransformType::WEIGHT, KernelSize, TransformSize, T> transform;
			for (int row = 0; row < TileSize; row++)
			{
				Line<T, KernelSize> line;
				for (int col = 0; col < KernelSize; col++)
					line[col] = transform(row, tile.get_row(col)); // tile is stored as transposed (column-major)

				for (int col = 0; col < TileSize; col++)
					matrices[matrices_indexer.at(row, col, blockIdx.y, f)] = transform(col, line);
			}
		}
	}

	template<int KernelSize, int TransformSize, typename T>
	__global__ void kernel_transform_input(T *__restrict__ matrices, const T *__restrict__ input, int batch_size, int height, int width,
			int input_filters)
	{
		constexpr int TileSize = KernelSize + TransformSize - 1;
		constexpr int Padding = KernelSize / 2;

		Tile<T, TileSize, TileSize> tile;
		for (int f = threadIdx.x; f < input_filters; f += blockDim.x)
		{
			const Indexer<4> input_indexer(batch_size, height, width, input_filters);
			for (int col = 0; col < tile.columns(); col++)
				for (int row = 0; row < tile.rows(); row++)
				{
					const int h = TransformSize * blockIdx.x - Padding + row;
					const int w = TransformSize * blockIdx.y - Padding + col;
					tile.at(col, row) = is_inside(h, w, height, width) ? input[input_indexer.at(blockIdx.z, h, w, f)] : T(0.0f);
				}

			const int tile_index = (blockIdx.z * gridDim.x + blockIdx.x) * gridDim.y + blockIdx.y;

			const Indexer<4> matrices_indexer(TileSize, TileSize, gridDim.x * gridDim.y * gridDim.z, input_filters);
			const Transform<TransformType::INPUT, KernelSize, TransformSize, T> transform;
			for (int row = 0; row < TileSize; row++)
			{
				Line<T, TileSize> line;
				for (int col = 0; col < TileSize; col++)
					line[col] = transform(row, tile.get_row(col)); // tile is stored as transposed (column-major)

				for (int col = 0; col < TileSize; col++)
					matrices[matrices_indexer.at(row, col, tile_index, f)] = transform(col, line);
			}
		}
	}
	template<int KernelSize, int TransformSize, typename T>
	__global__ void kernel_transform_output(const T *__restrict__ matrices, T *__restrict__ output, const T *__restrict__ add,
			const T *__restrict__ bias, mlActivationType_t activation, int batch_size, int height, int width, int output_filters)
	{
		constexpr int TileSize = KernelSize + TransformSize - 1;

		Tile<T, TileSize, TileSize> tile;
		for (int f = threadIdx.x; f < output_filters; f += blockDim.x)
		{
			const T bias_value = (bias != nullptr) ? bias[f] : T(0.0f);

			const int tile_index = (blockIdx.z * gridDim.x + blockIdx.x) * gridDim.y + blockIdx.y;
			const Indexer<4> matrices_indexer(TileSize, TileSize, gridDim.x * gridDim.y * gridDim.z, output_filters);
			for (int col = 0; col < tile.columns(); col++)
				for (int row = 0; row < tile.rows(); row++)
					tile.at(col, row) = matrices[matrices_indexer.at(row, col, tile_index, f)];

			const Indexer<4> output_indexer(batch_size, height, width, output_filters);
			const Transform<TransformType::OUTPUT, KernelSize, TransformSize, T> transform;
			for (int row = 0; row < TransformSize; row++)
			{
				const int h = TransformSize * blockIdx.x + row;
				if (h < height)
				{
					Line<T, TileSize> line;
					for (int col = 0; col < TileSize; col++)
						line[col] = transform(row, tile.get_row(col));

					for (int col = 0; col < TransformSize; col++)
					{
						const int w = TransformSize * blockIdx.y + col;
						if (w < width)
						{
							T tmp = transform(col, line);

							if (add != nullptr)
								tmp += add[output_indexer.at(blockIdx.z, h, w, f)];
							if (bias != nullptr)
								tmp += bias_value;
							switch (activation)
							{
								case ACTIVATION_SIGMOID:
									tmp = vectors::sigmoid(tmp);
									break;
								case ACTIVATION_TANH:
									tmp = vectors::tanh(tmp);
									break;
								case ACTIVATION_RELU:
									tmp = vectors::relu(tmp);
									break;
								case ACTIVATION_LEAKY_RELU:
									tmp = vectors::leaky_relu(tmp);
									break;
							}

							output[output_indexer.at(blockIdx.z, h, w, f)] = tmp;
						}
					}
				}
			}
		}
	}

	template<int KernelSize, int TransformSize, typename T>
	__global__ void kernel_transform_gradient(T *__restrict__ matrices, const T *__restrict__ gradient, int batch_size, int height, int width,
			int output_filters)
	{
		constexpr int TileSize = KernelSize + TransformSize - 1;

		Tile<T, TransformSize, TransformSize> tile;
		for (int f = threadIdx.x; f < output_filters; f += blockDim.x)
		{
			const Indexer<4> gradient_indexer(batch_size, height, width, output_filters);
			for (int col = 0; col < tile.columns(); col++)
				for (int row = 0; row < tile.rows(); row++)
				{
					const int h = TransformSize * blockIdx.x + row;
					const int w = TransformSize * blockIdx.y + col;
					tile.at(col, row) = is_inside(h, w, height, width) ? gradient[gradient_indexer.at(blockIdx.z, h, w, f)] : T(0.0f);
				}

			const int tile_index = (blockIdx.z * gridDim.x + blockIdx.x) * gridDim.y + blockIdx.y;
			const Indexer<4> matrices_indexer(TileSize, TileSize, gridDim.x * gridDim.y * gridDim.z, output_filters);
			const Transform<TransformType::GRADIENT, KernelSize, TransformSize, T> transform;
			for (int row = 0; row < TileSize; row++)
			{
				Line<T, TransformSize> line;
				for (int col = 0; col < TransformSize; col++)
					line[col] = transform(row, tile.get_row(col));

				for (int col = 0; col < TileSize; col++)
					matrices[matrices_indexer.at(row, col, tile_index, f)] = transform(col, line);
			}
		}
	}

	template<int KernelSize, int TransformSize, typename T, typename U>
	__global__ void kernel_transform_update(const T *__restrict__ matrices, U *__restrict__ update, int output_filters, int input_filters)
	{
		constexpr int TileSize = KernelSize + TransformSize - 1;

		Tile<T, TileSize, TileSize> tile;
		for (int f = threadIdx.x; f < input_filters; f += blockDim.x)
		{
			const Indexer<4> matrices_indexer(TileSize, TileSize, output_filters, input_filters);
			for (int col = 0; col < tile.columns(); col++)
				for (int row = 0; row < tile.rows(); row++)
					tile.at(col, row) = matrices[matrices_indexer.at(row, col, blockIdx.y, f)];

			const Indexer<4> update_indexer(output_filters, KernelSize, KernelSize, input_filters);
			const Transform<TransformType::UPDATE, KernelSize, TransformSize, T> transform;
			for (int row = 0; row < KernelSize; row++)
			{
				Line<T, TileSize> line;
				for (int col = 0; col < TileSize; col++)
					line[col] = transform(row, tile.get_row(col));

				for (int col = 0; col < KernelSize; col++)
					update[update_indexer.at(blockIdx.y, row, col, f)] = U(to_fp32(transform(col, line)));
			}
		}
	}

	/*
	 * host code
	 */
	template<typename T>
	int vector_length()
	{
		return 1;
	}
	template<>
	int vector_length<vec2h>()
	{
		return 2;
	}
	int get_number_of_tiles(int dim, int transform_size)
	{
		return (dim + transform_size - 1) / transform_size;
	}
	int get_kernel_size(const mlTensor_t &matrices, int tile_size)
	{
		assert(matrices.rank == 3);
		return std::sqrt(matrices.dim[0]) + 1 - tile_size;
	}

	template<typename T>
	void launch_weight_transform(mlContext_t context, int tile_size, const mlTensor_t w, mlTensor_t matrices, bool invert)
	{
		const int filters_out = w.dim[0];
		const int filters_in = w.dim[3] / vector_length<T>();

		const int kernel_size = get_kernel_size(matrices, tile_size);

		dim3 blockSize(std::min(128, filters_in));
		dim3 gridSize(1, filters_out);
		hipStream_t stream = ml::cuda_backend::Context::getStream(context);

		if (kernel_size == 3)
		{
			if (tile_size == 2)
				kernel_transform_weights<3, 2> <<<gridSize, blockSize, 0, stream>>>(data<T>(matrices), data<T>(w), filters_out, filters_in, invert);
			if (tile_size == 4)
				kernel_transform_weights<3, 4> <<<gridSize, blockSize, 0, stream>>>(data<T>(matrices), data<T>(w), filters_out, filters_in, invert);
		}
		if (kernel_size == 5 && tile_size == 2)
		{
			kernel_transform_weights<5, 2> <<<gridSize, blockSize, 0, stream>>>(data<T>(matrices), data<T>(w), filters_out, filters_in, invert);
		}
		assert(hipGetLastError() == hipSuccess);
	}
	template<typename T>
	void launch_input_transform(mlContext_t context, int tile_size, const mlTensor_t x, mlTensor_t matrices)
	{
		const int batch_size = x.dim[0];
		const int height = x.dim[1];
		const int width = x.dim[2];
		const int filters = x.dim[3] / vector_length<T>();

		const int kernel_size = get_kernel_size(matrices, tile_size);

		const int tiles_h = get_number_of_tiles(height, tile_size);
		const int tiles_w = get_number_of_tiles(width, tile_size);
		hipStream_t stream = ml::cuda_backend::Context::getStream(context);

		dim3 blockSize(std::min(128, filters));
		dim3 gridSize(tiles_h, tiles_w, x.dim[0]);

		if (kernel_size == 3)
		{
			if (tile_size == 2)
				kernel_transform_input<3, 2> <<<gridSize, blockSize, 0, stream>>>(data<T>(matrices), data<T>(x), batch_size, height, width, filters);
			if (tile_size == 4)
				kernel_transform_input<3, 4> <<<gridSize, blockSize, 0, stream>>>(data<T>(matrices), data<T>(x), batch_size, height, width, filters);
		}
		if (kernel_size == 5 && tile_size == 2)
		{
			kernel_transform_input<5, 2> <<<gridSize, blockSize, 0, stream>>>(data<T>(matrices), data<T>(x), batch_size, height, width, filters);
		}

		assert(hipGetLastError() == hipSuccess);
	}
	template<typename T>
	void launch_output_transform(mlContext_t context, int tile_size, const mlTensor_t matrices, const mlTensor_t bias, const mlTensor_t ext,
			mlTensor_t y, mlActivationType_t act)
	{
		const int batch_size = y.dim[0];
		const int height = y.dim[1];
		const int width = y.dim[2];
		const int filters = y.dim[3] / vector_length<T>();

		const int kernel_size = get_kernel_size(matrices, tile_size);

		const int tiles_h = get_number_of_tiles(height, tile_size);
		const int tiles_w = get_number_of_tiles(width, tile_size);
		hipStream_t stream = ml::cuda_backend::Context::getStream(context);

		dim3 blockSize(std::min(128, filters));
		dim3 gridSize(tiles_h, tiles_w, batch_size);

		if (kernel_size == 3)
		{
			if (tile_size == 2)
				kernel_transform_output<3, 2> <<<gridSize, blockSize, 0, stream>>>(data<T>(matrices), data<T>(y), data<T>(ext), data<T>(bias), act,
						batch_size, height, width, filters);
			if (tile_size == 4)
				kernel_transform_output<3, 4> <<<gridSize, blockSize, 0, stream>>>(data<T>(matrices), data<T>(y), data<T>(ext), data<T>(bias), act,
						batch_size, height, width, filters);
		}
		if (kernel_size == 5 && tile_size == 2)
		{
			kernel_transform_output<5, 2> <<<gridSize, blockSize, 0, stream>>>(data<T>(matrices), data<T>(y), data<T>(ext), data<T>(bias), act,
					batch_size, height, width, filters);
		}

		assert(hipGetLastError() == hipSuccess);
	}
	template<typename T>
	void launch_gradient_transform(mlContext_t context, int tile_size, const mlTensor_t dy, mlTensor_t matrices)
	{
		const int batch_size = dy.dim[0];
		const int height = dy.dim[1];
		const int width = dy.dim[2];
		const int filters = dy.dim[3] / vector_length<T>();

		const int kernel_size = get_kernel_size(matrices, tile_size);

		const int tiles_h = get_number_of_tiles(height, tile_size);
		const int tiles_w = get_number_of_tiles(width, tile_size);
		hipStream_t stream = ml::cuda_backend::Context::getStream(context);

		dim3 blockSize(std::min(128, filters));
		dim3 gridSize(tiles_h, tiles_w, dy.dim[0]);

		if (kernel_size == 3)
		{
			if (tile_size == 2)
				kernel_transform_gradient<3, 2> <<<gridSize, blockSize, 0, stream>>>(data<T>(matrices), data<T>(dy), batch_size, height, width,
						filters);
			if (tile_size == 4)
				kernel_transform_gradient<3, 4> <<<gridSize, blockSize, 0, stream>>>(data<T>(matrices), data<T>(dy), batch_size, height, width,
						filters);
		}
		if (kernel_size == 5 && tile_size == 2)
		{
			kernel_transform_gradient<5, 2> <<<gridSize, blockSize, 0, stream>>>(data<T>(matrices), data<T>(dy), batch_size, height, width, filters);
		}
		assert(hipGetLastError() == hipSuccess);
	}
	template<typename T, typename U>
	void launch_update_transform(mlContext_t context, int tile_size, const mlTensor_t matrices, mlTensor_t dw)
	{
		const int filters_out = dw.dim[0];
		const int filters_in = dw.dim[3] / vector_length<T>();

		const int kernel_size = get_kernel_size(matrices, tile_size);

		dim3 blockSize(std::min(128, filters_in));
		dim3 gridSize(1, filters_out);
		hipStream_t stream = ml::cuda_backend::Context::getStream(context);

		if (kernel_size == 3)
		{
			if (tile_size == 2)
				kernel_transform_update<3, 2> <<<gridSize, blockSize, 0, stream>>>(data<T>(matrices), data<U>(dw), filters_out, filters_in);
			if (tile_size == 4)
				kernel_transform_update<3, 4> <<<gridSize, blockSize, 0, stream>>>(data<T>(matrices), data<U>(dw), filters_out, filters_in);
		}
		if (kernel_size == 5 && tile_size == 2)
		{
			kernel_transform_update<5, 2> <<<gridSize, blockSize, 0, stream>>>(data<T>(matrices), data<U>(dw), filters_out, filters_in);
		}

		assert(hipGetLastError() == hipSuccess);
	}
}

namespace ml
{
	void cuda_winograd_weight_transform(mlContext_t context, int tile_size, const mlTensor_t w, mlTensor_t matrices, bool invert)
	{
		assert(w.dtype == matrices.dtype);
		switch (matrices.dtype)
		{
			case DTYPE_FLOAT16:
			{
				if (get_last_dim(w) % 2 == 0)
					launch_weight_transform<vec2h>(context, tile_size, w, matrices, invert);
				else
					launch_weight_transform<vec1h>(context, tile_size, w, matrices, invert);
				break;
			}
			case DTYPE_FLOAT32:
				launch_weight_transform<vec1f>(context, tile_size, w, matrices, invert);
				break;
		}
	}
	void cuda_winograd_input_transform(mlContext_t context, int tile_size, const mlTensor_t x, mlTensor_t matrices)
	{
		assert(x.dtype == matrices.dtype);
		switch (matrices.dtype)
		{
			case DTYPE_FLOAT16:
			{
				if (get_last_dim(x) % 2 == 0)
					launch_input_transform<vec2h>(context, tile_size, x, matrices);
				else
					launch_input_transform<vec1h>(context, tile_size, x, matrices);
				break;
			}
			case DTYPE_FLOAT32:
				launch_input_transform<vec1f>(context, tile_size, x, matrices);
				break;
		}
	}
	void cuda_winograd_output_transform(mlContext_t context, int tile_size, const mlTensor_t matrices, const mlTensor_t bias, const mlTensor_t ext,
			mlTensor_t y, mlActivationType_t act)
	{
		assert(matrices.dtype == y.dtype);
		switch (matrices.dtype)
		{
			case DTYPE_FLOAT16:
			{
				if (get_last_dim(y) % 2 == 0)
					launch_output_transform<vec2h>(context, tile_size, matrices, bias, ext, y, act);
				else
					launch_output_transform<vec1h>(context, tile_size, matrices, bias, ext, y, act);
				break;
			}
			case DTYPE_FLOAT32:
				launch_output_transform<vec1f>(context, tile_size, matrices, bias, ext, y, act);
				break;
		}
	}
	void cuda_winograd_gradient_transform(mlContext_t context, int tile_size, const mlTensor_t dy, mlTensor_t matrices)
	{
		assert(matrices.dtype == dy.dtype);
		switch (matrices.dtype)
		{
			case DTYPE_FLOAT16:
			{
				if (get_last_dim(dy) % 2 == 0)
					launch_gradient_transform<vec2h>(context, tile_size, dy, matrices);
				else
					launch_gradient_transform<vec1h>(context, tile_size, dy, matrices);
				break;
			}
			case DTYPE_FLOAT32:
				launch_gradient_transform<vec1f>(context, tile_size, dy, matrices);
				break;
		}
	}
	void cuda_winograd_update_transform(mlContext_t context, int tile_size, const mlTensor_t matrices, mlTensor_t dw)
	{
		switch (matrices.dtype)
		{
			case DTYPE_FLOAT16:
			{
				assert(is_fp32(dw) || is_fp16(dw));
				if (is_fp16(dw))
					launch_update_transform<vec1h, vec1h>(context, tile_size, matrices, dw);
				else
					launch_update_transform<vec1h, vec1f>(context, tile_size, matrices, dw);
				break;
			}
			case DTYPE_FLOAT32:
			{
				assert(is_fp32(dw));
				launch_update_transform<vec1f, vec1f>(context, tile_size, matrices, dw);
				break;
			}
		}
	}
} /* namespace ml */

