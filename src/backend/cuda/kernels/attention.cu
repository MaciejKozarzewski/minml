#include "hip/hip_runtime.h"
/*
 * attention.cpp
 *
 *  Created on: Jun 13, 2024
 *      Author: Maciej Kozarzewski
 */

#include <minml/backend/cuda_backend.h>
#include <minml/backend/backend_utils.hpp>

#include "../utils.hpp"
#include "../helpers/indexers.cuh"
#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#include <cinttypes>
#include <iostream>

namespace
{
	template<typename T>
	__device__ T clamp(T x, T lower, T upper)
	{
		assert(lower <= upper);
		return max(lower, min(upper, x));
	}
	__device__ void* apply_offset(void *ptr, int offsetInBytes)
	{
		return reinterpret_cast<uint8_t*>(ptr) + offsetInBytes;
	}
	__global__ void kernel_calculate_pointers(void *q_ptr[], void *k_ptr[], void *v_ptr[], void *input, void *qk_ptr[], void *workspace,
			void *out_ptr[], void *output, int batch_size, int tokens, int num_heads, int head_dim, int dtype_size)
	{
		const Indexer<5> input_indexer(batch_size, tokens, 3, num_heads, head_dim);
		const Indexer<4> workspace_indexer(batch_size, num_heads, tokens, tokens);
		const Indexer<4> output_indexer(batch_size, tokens, num_heads, head_dim);

		for (int i = threadIdx.x; i < batch_size * num_heads; i += blockDim.x)
		{
			const int idx_b = i / num_heads;
			const int idx_h = i % num_heads;

			q_ptr[i] = apply_offset(input, dtype_size * input_indexer.at(idx_b, 0, 0, idx_h, 0));
			k_ptr[i] = apply_offset(input, dtype_size * input_indexer.at(idx_b, 0, 1, idx_h, 0));
			v_ptr[i] = apply_offset(input, dtype_size * input_indexer.at(idx_b, 0, 2, idx_h, 0));
			qk_ptr[i] = apply_offset(workspace, dtype_size * workspace_indexer.at(idx_b, idx_h, 0, 0));
			out_ptr[i] = apply_offset(output, dtype_size * output_indexer.at(idx_b, 0, idx_h, 0));
		}
	}

	template<typename T>
	__global__ void kernel_softmax_forward_in_place(T *input, const T *weights, int batch_size, int num_heads, int height, int width, int range)
	{
		assert(height * width <= 512);
		assert(blockDim.x == 128);
		__shared__ float workspace[512];
		__shared__ int2 indices[512];
		__shared__ cg::block_tile_memory<128> btm;
		cg::thread_block thb = cg::this_thread_block(btm);
		cg::thread_block_tile<128> tile = cg::tiled_partition<128>(thb);

		const int batch_idx = blockIdx.y;
		const int head_idx = blockIdx.z;
		const int tokens = height * width;

		const Indexer<3> weight_indexer(num_heads, 2 * range + 1, 2 * range + 1);
		const Indexer<4> input_indexer(batch_size, num_heads, tokens, tokens);

		for (int i = threadIdx.x; i < height * width; i += blockDim.x)
		{
			indices[i].x = i / width;
			indices[i].y = i % width;
		}
		__syncthreads();

		for (int i = blockIdx.x; i < tokens; i += gridDim.x)
		{
			const int2 origin = indices[i];

			const int idx = input_indexer.at(batch_idx, head_idx, i, 0);
			float max_value = -1e+32f;
			for (int j = tile.thread_rank(); j < tokens; j += tile.size())
			{
				const int2 current = indices[j];
				const int offset_x = range + clamp(current.x - origin.x, -range, range);
				const int offset_y = range + clamp(current.y - origin.y, -range, range);
				const float bias = static_cast<float>(weights[weight_indexer.at(head_idx, offset_x, offset_y)]);
				workspace[j] = bias + static_cast<float>(input[idx + j]);
				max_value = max(max_value, workspace[j]);
			}
			const float shift = cg::reduce(tile, max_value, cg::greater<float>());

			float partial_sum = 0.0f;
			for (int j = tile.thread_rank(); j < tokens; j += tile.size())
			{
				workspace[j] = exp(workspace[j] - shift);
				partial_sum += workspace[j];
			}
			const float inv_sum = 1.0f / cg::reduce(tile, partial_sum, cg::plus<float>());
			for (int j = tile.thread_rank(); j < tokens; j += tile.size())
				input[idx + j] = static_cast<T>(workspace[j] * inv_sum);
		}
	}
	__global__ void kernel_softmax_backward_in_place(const float *output, float *gradient, float *weights_update, int batch_size, int num_heads,
			int height, int width, int range)
	{
		assert(height * width <= 512);
		__shared__ int2 indices[512];
		__shared__ float workspace[2048];

		const int batch_idx = blockIdx.x;
		const int head_idx = blockIdx.y;
		const int tokens = height * width;
		const int size = 2 * range + 1;

		for (int i = threadIdx.x; i < tokens; i += blockDim.x)
		{
			indices[i].x = i / width;
			indices[i].y = i % width;
		}
		assert(size * size <= 2048);
		for (int i = threadIdx.x; i < size * size; i += blockDim.x)
			workspace[i] = 0.0f;
		__syncthreads();

		const Indexer<4> gradient_indexer(batch_size, num_heads, tokens, tokens);
		for (int i = 0; i < tokens; i++)
		{
			const int2 origin = indices[i];

			const int idx = gradient_indexer.at(batch_idx, head_idx, i, 0);
			for (int j = threadIdx.x; j < tokens; j += blockDim.x)
			{
				const int2 current = indices[j];
				const int offset_h = range + clamp(current.x - origin.x, -range, range);
				const int offset_w = range + clamp(current.y - origin.y, -range, range);
				const float out = output[idx + j];
				const float grad = gradient[idx + j] * out * (1.0f - out);

				atomicAdd(workspace + offset_h * size + offset_w, grad);
				gradient[idx + j] = grad;
			}
			__syncthreads();
		}

		const Indexer<3> update_indexer(batch_size, num_heads, size * size);
		for (int i = threadIdx.x; i < size * size; i += blockDim.x)
			weights_update[update_indexer.at(batch_idx, head_idx, i)] += workspace[i];
	}
	__global__ void kernel_weights_update_reduction(const float *workspace, float *update, int batch_size, int num_heads, int last_dim)
	{
		assert(blockDim.x == 32);
		assert(blockDim.y == 32);
		__shared__ float storage[32 * 32];

		const Indexer<3> workspace_indexer(batch_size, num_heads, last_dim);

		const int tid = blockIdx.x * 32 + threadIdx.x;
		float d_w = 0.0f;
		if (tid < last_dim)
			for (int i = threadIdx.y; i < batch_size; i += 32)
				d_w += workspace[workspace_indexer.at(i, blockIdx.y, tid)];
		storage[threadIdx.y * 32 + threadIdx.x] = d_w;

		__syncthreads();
		assert(blockDim.x == 32 && blockDim.y == 32);
		for (int i = 16; i >= 1; i /= 2)
		{
			if (threadIdx.y < i)
				storage[threadIdx.y * 32 + threadIdx.x] += storage[(i + threadIdx.y) * 32 + threadIdx.x];
			__syncthreads();
		}

		const Indexer<2> update_indexer(num_heads, last_dim);
		if (threadIdx.y == 0 && tid < last_dim)
			update[update_indexer.at(blockIdx.y, tid)] += storage[threadIdx.x];
	}

	void gemm_batched(ml::mlContext_t context, char opA, char opB, ml::mlDataType_t dtype, int M, int N, int K, float alpha, const void *A[], int lda,
			const void *B[], int ldb, float beta, void *C[], int ldc, int batch_count)
	{
		hipblasOperation_t transa = ml::is_transpose(opA) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
		hipblasOperation_t transb = ml::is_transpose(opB) ? HIPBLAS_OP_T : HIPBLAS_OP_N;

		hipblasHandle_t handle = ml::cuda::Context::getHandle(context);
		hipblasStatus_t err = hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH);
		assert(err == HIPBLAS_STATUS_SUCCESS);
		switch (dtype)
		{
			case ml::DTYPE_FLOAT16:
			{
				if (ml::cuda::has_fp16_math(context))
				{
					const half _alpha = alpha;
					const half _beta = beta;
					hipblasStatus_t status = hipblasHgemmBatched(handle, transb, transa, N, M, K, &_alpha, ml::getPointer<half*>(B), ldb,
							ml::getPointer<half*>(A), lda, &_beta, ml::getPointer<half*>(C), ldc, batch_count);
					assert(status == HIPBLAS_STATUS_SUCCESS);
					break;
				}
				else
				{
					const float _alpha = alpha;
					const float _beta = beta;
					hipblasStatus_t status = hipblasGemmBatchedEx(handle, transb, transa, N, M, K, &_alpha, ml::getPointer<void*>(B), HIP_R_16F, ldb,
							ml::getPointer<void*>(A), HIP_R_16F, lda, &_beta, ml::getPointer<void*>(C), HIP_R_16F, ldc, batch_count,
							HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT);
					assert(status == HIPBLAS_STATUS_SUCCESS);
					break;
				}
			}
			case ml::DTYPE_FLOAT32:
			{
				const float _alpha = alpha;
				const float _beta = beta;
				hipblasStatus_t status = hipblasSgemmBatched(handle, transb, transa, N, M, K, &_alpha, ml::getPointer<float*>(B), ldb,
						ml::getPointer<float*>(A), lda, &_beta, ml::getPointer<float*>(C), ldc, batch_count);
				assert(status == HIPBLAS_STATUS_SUCCESS);
				break;
			}
		}
	}
	void run_softmax_forward(hipStream_t stream, void *input, ml::mlShape_t input_shape, const void *weights, ml::mlShape_t weights_shape,
			ml::mlDataType_t dtype)
	{
		const int batch_size = input_shape.dim[0];
		const int height = input_shape.dim[1];
		const int width = input_shape.dim[2];
		const int num_heads = weights_shape.dim[0];
		assert(weights_shape.dim[1] == weights_shape.dim[2]);
		const int range = (weights_shape.dim[1] - 1) / 2;

		dim3 blockDim(128);
		dim3 gridDim(32, batch_size, num_heads);

		switch (dtype)
		{
			case ml::DTYPE_FLOAT16:
			{
				kernel_softmax_forward_in_place<<<gridDim, blockDim, 0, stream>>>(ml::getPointer<half>(input), ml::getPointer<half>(weights),
						batch_size, num_heads, height, width, range);
				break;
			}
			case ml::DTYPE_FLOAT32:
			{
				kernel_softmax_forward_in_place<<<gridDim, blockDim, 0, stream>>>(ml::getPointer<float>(input), ml::getPointer<float>(weights),
						batch_size, num_heads, height, width, range);
				break;
			}
		}
		assert(hipGetLastError() == hipSuccess);
	}
}

namespace ml
{
	int cuda_multi_head_attention_get_workspace_size(mlShape_t input_shape, mlShape_t weights_shape, bool training)
	{
		assert(input_shape.rank == 4);
		assert(weights_shape.rank == 3);
		const int batch_size = input_shape.dim[0];
		const int tokens = input_shape.dim[1] * input_shape.dim[2];
		const int num_heads = weights_shape.dim[0];

		int result = batch_size * num_heads * tokens * tokens;
		if (training)
			result = result * 2 + batch_size * num_heads * weights_shape.dim[1] * weights_shape.dim[2];
		return result;
	}
	void cuda_multi_head_attention_forward(mlContext_t context, mlShape_t input_shape, mlShape_t weights_shape, mlDataType_t dtype, const void *input,
			void *output, const void *weights, void *workspace)
	{
		assert(input_shape.rank == 4);
		assert(weights_shape.rank == 3);
		const int batch_size = input_shape.dim[0];
		const int height = input_shape.dim[1];
		const int width = input_shape.dim[2];
		const int tokens = height * width;
		const int embedding = input_shape.dim[3] / 3;
		const int num_heads = weights_shape.dim[0];
		const int head_dim = embedding / num_heads;

		const int num_pointers = batch_size * num_heads;
		void **pointers = getPointer<void*>(cuda::Context::getWorkspace(context));

		void **q_ptr = pointers + 0 * num_pointers;
		void **k_ptr = pointers + 1 * num_pointers;
		void **v_ptr = pointers + 2 * num_pointers;
		void **qk_ptr = pointers + 3 * num_pointers;
		void **out_ptr = pointers + 4 * num_pointers;

		hipStream_t stream = cuda::Context::getStream(context);

		kernel_calculate_pointers<<<1, 1024, 0, stream>>>(q_ptr, k_ptr, v_ptr, const_cast<void*>(input), qk_ptr, workspace, out_ptr, output,
				batch_size, tokens, num_heads, head_dim, size_of(dtype));
		assert(hipGetLastError() == hipSuccess);

		const float scale = 1.0f / std::sqrt(head_dim);
		gemm_batched(context, 'n', 't', dtype, tokens, tokens, head_dim, scale, const_cast<const void**>(q_ptr), 3 * embedding,
				const_cast<const void**>(k_ptr), 3 * embedding, 0.0f, qk_ptr, tokens, num_pointers);

		run_softmax_forward(stream, workspace, input_shape, weights, weights_shape, dtype);

		gemm_batched(context, 'n', 'n', dtype, tokens, head_dim, tokens, 1.0f, const_cast<const void**>(qk_ptr), tokens,
				const_cast<const void**>(v_ptr), 3 * embedding, 0.0f, out_ptr, embedding, num_pointers);
		assert(hipGetLastError() == hipSuccess);
	}
	void cuda_multi_head_attention_backward(mlContext_t context, mlShape_t input_shape, mlShape_t weights_shape, const void *input,
			const void *weights, void *gradient_prev, void *gradient_next, void *weights_update, void *workspace)
	{
		assert(input_shape.rank == 4);
		assert(weights_shape.rank == 3);
		const int batch_size = input_shape.dim[0];
		const int height = input_shape.dim[1];
		const int width = input_shape.dim[2];
		const int tokens = height * width;
		const int embedding = input_shape.dim[3] / 3;
		const int num_heads = weights_shape.dim[0];
		const int head_dim = embedding / num_heads;
		assert(weights_shape.dim[1] == weights_shape.dim[2]);
		const int range = (weights_shape.dim[1] - 1) / 2;

		const int offset = batch_size * num_heads * tokens * tokens * size_of(DTYPE_FLOAT32);
		void *forward_workspace = workspace;
		void *backward_workspace = reinterpret_cast<void*>(reinterpret_cast<uint8_t*>(workspace) + offset);
		void *update_workspace = reinterpret_cast<void*>(reinterpret_cast<uint8_t*>(workspace) + 2 * offset);

		const int num_pointers = batch_size * num_heads;
		void **pointers = getPointer<void*>(cuda::Context::getWorkspace(context));

		void **q_ptr = pointers + 0 * num_pointers;
		void **k_ptr = pointers + 1 * num_pointers;
		void **v_ptr = pointers + 2 * num_pointers;
		void **qk_ptr = pointers + 3 * num_pointers;
		void **out_ptr = pointers + 4 * num_pointers;

		void **dq_ptr = pointers + 5 * num_pointers;
		void **dk_ptr = pointers + 6 * num_pointers;
		void **dv_ptr = pointers + 7 * num_pointers;
		void **dqk_ptr = pointers + 8 * num_pointers;
		void **dout_ptr = pointers + 9 * num_pointers;

		hipStream_t stream = cuda::Context::getStream(context);

		kernel_calculate_pointers<<<1, 1024, 0, stream>>>(q_ptr, k_ptr, v_ptr, const_cast<void*>(input), qk_ptr, forward_workspace, out_ptr, nullptr,
				batch_size, tokens, num_heads, head_dim, size_of(DTYPE_FLOAT32));
		kernel_calculate_pointers<<<1, 1024, 0, stream>>>(dq_ptr, dk_ptr, dv_ptr, gradient_prev, dqk_ptr, backward_workspace, dout_ptr, gradient_next,
				batch_size, tokens, num_heads, head_dim, size_of(DTYPE_FLOAT32));

		const float scale = 1.0f / std::sqrt(head_dim);
		gemm_batched(context, 'n', 't', DTYPE_FLOAT32, tokens, tokens, head_dim, scale, const_cast<const void**>(q_ptr), 3 * embedding,
				const_cast<const void**>(k_ptr), 3 * embedding, 0.0f, qk_ptr, tokens, num_pointers);

		run_softmax_forward(stream, forward_workspace, input_shape, weights, weights_shape, DTYPE_FLOAT32);

		// dqk = dy * V^T
		gemm_batched(context, 'n', 't', DTYPE_FLOAT32, tokens, tokens, head_dim, 1.0f, const_cast<const void**>(dout_ptr), embedding,
				const_cast<const void**>(v_ptr), 3 * embedding, 0.0f, dqk_ptr, tokens, num_pointers);
		// dV = qk^T * dy
		gemm_batched(context, 't', 'n', DTYPE_FLOAT32, tokens, head_dim, tokens, 1.0f, const_cast<const void**>(qk_ptr), tokens,
				const_cast<const void**>(dout_ptr), embedding, 0.0f, dv_ptr, 3 * embedding, num_pointers);

		dim3 blockDim(256);
		dim3 gridDim(batch_size, num_heads);
		kernel_softmax_backward_in_place<<<gridDim, blockDim, 0, stream>>>(getPointer<float>(forward_workspace),
				getPointer<float>(backward_workspace), getPointer<float>(update_workspace), batch_size, num_heads, height, width, range);
		assert(hipGetLastError() == hipSuccess);

		const int last_dim = weights_shape.dim[1] * weights_shape.dim[2];
		blockDim = dim3(32, 32);
		gridDim = dim3((last_dim + 31) / 32, num_heads);
		kernel_weights_update_reduction<<<gridDim, blockDim, 0, stream>>>(getPointer<float>(update_workspace), getPointer<float>(weights_update),
				batch_size, num_heads, last_dim);
		assert(hipGetLastError() == hipSuccess);

// dQ = dqk * K
		gemm_batched(context, 'n', 'n', DTYPE_FLOAT32, tokens, head_dim, tokens, 1.0f / scale, const_cast<const void**>(dqk_ptr), tokens,
				const_cast<const void**>(k_ptr), 3 * embedding, 0.0f, dq_ptr, 3 * embedding, num_pointers);
		// dK = dqk^T * Q
		gemm_batched(context, 't', 'n', DTYPE_FLOAT32, tokens, head_dim, tokens, 1.0f / scale, const_cast<const void**>(dqk_ptr), tokens,
				const_cast<const void**>(q_ptr), 3 * embedding, 0.0f, dk_ptr, 3 * embedding, num_pointers);

		assert(hipGetLastError() == hipSuccess);
	}
} /* namespace ml */

