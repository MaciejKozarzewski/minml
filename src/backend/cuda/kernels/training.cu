#include "hip/hip_runtime.h"
/*
 * training.cu
 *
 *  Created on: Jan 4, 2023
 *      Author: Maciej Kozarzewski
 */

#include <minml/backend/cuda_backend.h>
#include <minml/backend/backend_utils.hpp>

#include "../utils.hpp"
#include "../vectors/vectors.cuh"

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#include <cmath>
#include <iostream>
#include <cassert>
#include <vector>

namespace
{
	using namespace vectors;

	__device__ float round_small_to_zero(float x)
	{
		return (fabsf(x) < 1.0e-6f) ? 0.0f : x;
	}
	__device__ float safe_log(float x)
	{
		return logf(1.0e-8f + x);
	}
	__device__ float cross_entropy(float output, float target)
	{
		return -target * safe_log(output) - (1.0f - target) * safe_log(1.0f - output);
	}
	__device__ float square(float x)
	{
		return x * x;
	}

	__global__ void kernel_loss_gradient(float *gradient, const float *output, const float *target, int elements, float inv_batch_size)
	{
		for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < elements; i += gridDim.x * blockDim.x)
			gradient[i] = inv_batch_size * (output[i] - target[i]);
	}
	__global__ void kernel_loss_step_1(float *workspace, const float *output, const float *target, int elements)
	{
		assert(blockDim.x == 256);
		__shared__ cg::block_tile_memory<256> btm;
		cg::thread_block thb = cg::this_thread_block(btm);
		cg::thread_block_tile<256> tile = cg::tiled_partition<256>(thb);

		float acc = 0.0f;
		for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < elements; i += gridDim.x * blockDim.x)
			acc += max(0.0f, cross_entropy(output[i], target[i]) - cross_entropy(target[i], target[i]));
		const float sum = cg::reduce(tile, acc, cg::plus<float>());
		if (threadIdx.x == 0)
			workspace[blockIdx.x] = sum;
	}
	__global__ void kernel_loss_step_2(float *workspace, int elements)
	{
		assert(gridDim.x == 1);
		assert(blockDim.x == 256);
		__shared__ cg::block_tile_memory<256> btm;
		cg::thread_block thb = cg::this_thread_block(btm);
		cg::thread_block_tile<256> tile = cg::tiled_partition<256>(thb);

		float acc = 0.0f;
		for (int i = threadIdx.x; i < elements; i += blockDim.x)
			acc += workspace[i];
		const float sum = cg::reduce(tile, acc, cg::plus<float>());
		if (threadIdx.x == 0)
			workspace[0] = sum;
	}

	__global__ void kernel_learn_adam(float *weight, const float *gradient, float *momentum, float *variance, int elements, float learning_rate,
			float beta1, float beta2)
	{
		for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < elements; i += gridDim.x * blockDim.x)
		{
			momentum[i] = momentum[i] * beta1 + gradient[i] * (1.0f - beta1);
			variance[i] = variance[i] * beta2 + square(gradient[i]) * (1.0f - beta2);
			const float tmp = -momentum[i] * learning_rate / sqrt(variance[i] + 1.0e-8f);
			weight[i] = round_small_to_zero(weight[i] + tmp);
		}
	}

	__global__ void kernel_regularizer_l2(float *gradient, const float *param, float scale, float offset, int elements)
	{
		for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < elements; i += gridDim.x * blockDim.x)
			gradient[i] += scale * (param[i] - offset);
	}

	template<int step>
	__global__ void kernel_sum_over_first_dim(float *dst, const float *src, int first_dim, int last_dim, float beta)
	{
		__shared__ float tmp[32][32];

		const int tid = blockIdx.x * 32 + threadIdx.x;
		if (tid < last_dim)
		{
			float result = 0.0f;
			for (int i = 32 * blockIdx.y + threadIdx.y; i < first_dim; i += 32 * gridDim.y)
				result += src[i * last_dim + tid];
			tmp[threadIdx.y][threadIdx.x] = result;
		}
		__syncthreads();

		for (int i = 16; i >= 1; i /= 2) // sum results stored in temporary array
		{
			if (threadIdx.y < i)
				tmp[threadIdx.y][threadIdx.x] += tmp[i + threadIdx.y][threadIdx.x];
			__syncthreads();
		}

		__syncthreads();
		if (threadIdx.y == 0 && tid < last_dim)
		{
			if (step == 1) // write to temporary storage array
				dst[blockIdx.y * last_dim + tid] = tmp[0][threadIdx.x];
			if (step == 2) // write to final destination
			{
				if (beta == 0.0f)
					dst[tid] = tmp[0][threadIdx.x];
				else
					dst[tid] = beta * dst[tid] + tmp[0][threadIdx.x];
			}
		}
	}

	template<typename T>
	__global__ void kernel_add_tensors(T *dst, const T *src0, const T *src1, int elements)
	{
		for (int i = (blockIdx.x * blockDim.x + threadIdx.x) * vector_length<T>(); i < elements; i += gridDim.x * blockDim.x * vector_length<T>())
		{
			const int tmp = elements - i;
			const Vector<T> x0(src0 + i, tmp);
			const Vector<T> x1(src1 + i, tmp);
			const Vector<T> y = x0 + x1;
			y.store(dst + i, tmp);
		}
	}
	template<typename T>
	__global__ void kernel_add_tensors(T *dst, const T *src, int elements)
	{
		for (int i = (blockIdx.x * blockDim.x + threadIdx.x) * vector_length<T>(); i < elements; i += gridDim.x * blockDim.x * vector_length<T>())
		{
			const int tmp = elements - i;
			const Vector<T> x(src + i, tmp);
			const Vector<T> y = Vector<T>(dst + i, tmp) + x;
			y.store(dst + i, tmp);
		}
	}

	__global__ void kernel_emulate_low_precision(uint32_t *dst, const uint32_t *src, int elements)
	{
		for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < elements; i += gridDim.x * blockDim.x)
			dst[i] = src[i] & 0xFFFFF000u;
	}
}

namespace ml
{
	void cuda_emulate_low_precision(mlContext_t context, mlShape_t shape, void *dst, const void *src)
	{
		const int length = volume(shape);
		dim3 blockDim(256);
		dim3 gridDim = cuda::gridSize<1024>(length, blockDim.x);

		kernel_emulate_low_precision<<<gridDim, blockDim, 0, cuda::Context::getStream(context)>>>(getPointer<uint32_t>(dst),
				getPointer<uint32_t>(src), length);
		assert(hipGetLastError() == hipSuccess);
	}
	void cuda_add_tensors(mlContext_t context, mlDataType_t dtype, mlShape_t shape, void *dst, const void *src1, const void *src2)
	{
		assert(dst != nullptr);
		assert(src1 != nullptr);
		assert(src2 != nullptr);

		const int length = volume(shape);
		dim3 blockDim(256);
		dim3 gridDim = cuda::gridSize<1024>(length, blockDim.x);
		hipStream_t stream = cuda::Context::getStream(context);

		if (dst == src1)
		{ // in place addition
			switch (dtype)
			{
				case DTYPE_BFLOAT16:
					kernel_add_tensors<<<gridDim, blockDim, 0, stream>>>(getPointer<__hip_bfloat16 >(dst), getPointer<__hip_bfloat16 >(src2), length);
					break;
				case DTYPE_FLOAT16:
					kernel_add_tensors<<<gridDim, blockDim, 0, stream>>>(getPointer<half>(dst), getPointer<half>(src2), length);
					break;
				case DTYPE_FLOAT32:
					kernel_add_tensors<<<gridDim, blockDim, 0, stream>>>(getPointer<float>(dst), getPointer<float>(src2), length);
					break;
			}
		}
		else
		{
			switch (dtype)
			{
				case DTYPE_BFLOAT16:
					kernel_add_tensors<<<gridDim, blockDim, 0, stream>>>(getPointer<__hip_bfloat16 >(dst), getPointer<__hip_bfloat16 >(src1),
							getPointer<__hip_bfloat16 >(src2), length);
					break;
				case DTYPE_FLOAT16:
					kernel_add_tensors<<<gridDim, blockDim, 0, stream>>>(getPointer<half>(dst), getPointer<half>(src1), getPointer<half>(src2),
							length);
					break;
				case DTYPE_FLOAT32:
					kernel_add_tensors<<<gridDim, blockDim, 0, stream>>>(getPointer<float>(dst), getPointer<float>(src1), getPointer<float>(src2),
							length);
					break;
			}
		}
		assert(hipGetLastError() == hipSuccess);
	}
	void cuda_sum_over_first_dim(mlContext_t context, mlShape_t shape, void *dst, const void *src, float beta)
	{
		assert(dst != nullptr);
		assert(src != nullptr);

		const int first_dim = volume_without_last_dim(shape);
		const int last_dim = get_last_dim(shape);

		assert(cuda::Context::getWorkspaceSize(context) >= last_dim * sizeof(float));

		float *workspace = cuda::Context::getWorkspace<float>(context);
		const int workspace_first_dim = std::min((size_t) 256, cuda::Context::getWorkspaceSize(context) / (sizeof(float) * last_dim));

		dim3 blockDim(32, 32);
		dim3 gridDim1((last_dim + 31) / 32, workspace_first_dim);
		dim3 gridDim2((last_dim + 31) / 32);
		hipStream_t stream = cuda::Context::getStream(context);

		kernel_sum_over_first_dim<1> <<<gridDim1, blockDim, 0, stream>>>(workspace, getPointer<float>(src), first_dim, last_dim, beta);
		assert(hipGetLastError() == hipSuccess);
		kernel_sum_over_first_dim<2> <<<gridDim2, blockDim, 0, stream>>>(getPointer<float>(dst), workspace, workspace_first_dim, last_dim, beta);
		assert(hipGetLastError() == hipSuccess);
	}
	float cuda_cross_entropy_loss(mlContext_t context, mlShape_t shape, const void *output, const void *target)
	{
		assert(output != nullptr);
		assert(target != nullptr);

		const int length = volume(shape);

		assert(cuda::Context::getWorkspaceSize(context) >= 4096 * sizeof(float));

		float *workspace = cuda::Context::getWorkspace<float>(context);

		dim3 blockDim(256);
		dim3 gridDim = cuda::gridSize<4096>(length, blockDim.x);
		hipStream_t stream = cuda::Context::getStream(context);

		kernel_loss_step_1<<<gridDim, blockDim, 0, stream>>>(workspace, getPointer<float>(output), getPointer<float>(target), length);
		assert(hipGetLastError() == hipSuccess);

		kernel_loss_step_2<<<1, blockDim, 0, stream>>>(workspace, gridDim.x);
		assert(hipGetLastError() == hipSuccess);

		float result = 0.0f;
		hipMemcpyAsync(&result, workspace, sizeof(float), hipMemcpyDeviceToHost, stream);
		hipError_t status = hipStreamSynchronize(stream);
		assert(status == hipSuccess);
		return result / get_first_dim(shape);
	}
	void cuda_cross_entropy_gradient(mlContext_t context, mlShape_t shape, void *gradient, const void *output, const void *target, float weight)
	{
		assert(output != nullptr);
		assert(target != nullptr);
		assert(gradient != nullptr);

		const int length = volume(shape);
		const float inv_batch_size = weight / get_first_dim(shape);

		assert(cuda::Context::getWorkspaceSize(context) >= 4096 * sizeof(float));

		float *workspace = cuda::Context::getWorkspace<float>(context);

		dim3 blockDim(256);
		dim3 gridDim = cuda::gridSize<1024>(length, blockDim.x);
		hipStream_t stream = cuda::Context::getStream(context);

		kernel_loss_gradient<<<gridDim, blockDim, 0, stream>>>(getPointer<float>(gradient), getPointer<float>(output), getPointer<float>(target),
				length, inv_batch_size);
		assert(hipGetLastError() == hipSuccess);
	}
	void cuda_adam_optimize(mlContext_t context, mlShape_t shape, void *weight, const void *update, void *momentum, void *variance,
			float learning_rate, float beta1, float beta2)
	{
		assert(weight != nullptr);
		assert(update != nullptr);
		assert(momentum != nullptr);
		assert(variance != nullptr);
		const int length = volume(shape);
		dim3 blockDim(256);
		dim3 gridDim = cuda::gridSize<1024>(length, blockDim.x);
		hipStream_t stream = cuda::Context::getStream(context);

		kernel_learn_adam<<<gridDim, blockDim, 0, stream>>>(getPointer<float>(weight), getPointer<float>(update), getPointer<float>(momentum),
				getPointer<float>(variance), length, learning_rate, beta1, beta2);
		assert(hipGetLastError() == hipSuccess);
	}
	void cuda_l2_regularization(mlContext_t context, mlShape_t shape, void *gradient, const void *param, float coefficient, float offset)
	{
		assert(gradient != nullptr);
		assert(param != nullptr);

		const int length = volume(shape);
		dim3 blockDim(256);
		dim3 gridDim = cuda::gridSize<1024>(length, blockDim.x);

		kernel_regularizer_l2<<<gridDim, blockDim, 0, cuda::Context::getStream(context)>>>(getPointer<float>(gradient), getPointer<float>(param),
				coefficient, offset, length);
		assert(hipGetLastError() == hipSuccess);
	}
} /* namespace ml */

