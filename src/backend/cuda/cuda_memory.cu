#include "hip/hip_runtime.h"
/*
 * cuda_memory.cu
 *
 *  Created on: Jan 4, 2023
 *      Author: Maciej Kozarzewski
 */

#include <minml/backend/cuda_backend.h>
#include "utils.hpp"

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <cinttypes>
#include <cstring>
#include <iostream>
#include <cassert>

namespace
{
	using namespace ml::cuda;

	template<typename T>
	__global__ void kernel_setall(T *ptr, int length, T value)
	{
		for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < length; i += gridDim.x * blockDim.x)
			ptr[i] = value;
	}
	template<typename T>
	void setall_launcher(hipStream_t stream, void *dst, int dstSize, const void *value)
	{
		const int length = dstSize / sizeof(T);
		dim3 blockDim(256);
		dim3 gridDim = gridSize<1024>(length, blockDim.x);

		T v;
		std::memcpy(&v, value, sizeof(T));
		kernel_setall<<<gridDim, blockDim, 0, stream>>>(reinterpret_cast<T*>(dst), length, v);
		assert(hipGetLastError() == hipSuccess);

		if (stream == 0)
		{ // if launched from default stream this operation must be synchronous
			hipError_t status = hipDeviceSynchronize();
			assert(status == hipSuccess);
		}
	}

	bool is_aligned(const void *ptr, int bytes) noexcept
	{
		return (reinterpret_cast<std::uintptr_t>(ptr) % bytes) == 0;
	}
}

namespace ml
{
	void* cuda_malloc(int device_index, int count)
	{
		void *result = nullptr;
		if (count > 0)
		{
			hipError_t status = hipSetDevice(device_index);
			assert(status == hipSuccess);
			status = hipMalloc(reinterpret_cast<void**>(&result), count);
			assert(status == hipSuccess);
		}
		return result;
	}
	void cuda_page_lock(void *ptr, int count)
	{
		if (ptr != nullptr)
		{
			hipError_t status = hipHostRegister(ptr, count, 0);
			assert(status == hipSuccess);
		}
	}
	void cuda_page_unlock(void *ptr)
	{
		if (ptr != nullptr)
		{
			hipError_t status = hipHostUnregister(ptr);
			assert(status == hipSuccess);
		}
	}
	void cuda_free(void *ptr)
	{
		if (ptr != nullptr)
		{
			hipError_t status = hipFree(ptr);
			assert(status == hipSuccess);
		}
	}
	void* cuda_view(void *src, int offset, int count)
	{
		if (src == nullptr)
			return nullptr;
		else
			return reinterpret_cast<uint8_t*>(src) + offset;
	}

	void cuda_memset(mlContext_t context, void *dst, int dst_offset, int dst_count, const void *src, int src_count)
	{
		assert(dst != nullptr);
		void *tmp_dst = reinterpret_cast<uint8_t*>(dst) + dst_offset;
		if (src == nullptr)
		{
			if (context == nullptr)
			{
				cuda::Context::use(context);
				hipError_t status = hipMemset(tmp_dst, 0, dst_count);
				assert(status == hipSuccess);
				status = hipDeviceSynchronize(); // if launched from default stream this operation must be synchronous
				assert(status == hipSuccess);
			}
			else
			{
				hipError_t status = hipMemsetAsync(tmp_dst, 0, dst_count, cuda::Context::getStream(context));
				assert(status == hipSuccess);
			}
		}
		else
		{
			assert(dst_count % src_count == 0);
			assert(is_aligned(tmp_dst, src_count));

			switch (src_count)
			{
				case 2:
					setall_launcher<uint16_t>(cuda::Context::getStream(context), tmp_dst, dst_count, src);
					break;
				case 4:
					setall_launcher<uint32_t>(cuda::Context::getStream(context), tmp_dst, dst_count, src);
					break;
			}
		}
	}
	void cuda_memcpy_from_host(mlContext_t context, void *dst, int dst_offset, const void *src, int count)
	{
		assert(dst != nullptr);
		assert(src != nullptr);
		if (context == nullptr)
		{
			hipError_t status = hipMemcpy(reinterpret_cast<uint8_t*>(dst) + dst_offset, src, count, hipMemcpyHostToDevice);
			assert(status == hipSuccess);
		}
		else
		{
			hipError_t status = hipMemcpyAsync(reinterpret_cast<uint8_t*>(dst) + dst_offset, src, count, hipMemcpyHostToDevice,
					cuda::Context::getStream(context));
			assert(status == hipSuccess);

		}
	}
	void cuda_memcpy_to_host(mlContext_t context, void *dst, const void *src, int src_offset, int count)
	{
		assert(dst != nullptr);
		assert(src != nullptr);
		if (context == nullptr)
		{
			hipError_t status = hipMemcpy(dst, reinterpret_cast<const uint8_t*>(src) + src_offset, count, hipMemcpyDeviceToHost);
			assert(status == hipSuccess);
		}
		else
		{
			hipError_t status = hipMemcpyAsync(dst, reinterpret_cast<const uint8_t*>(src) + src_offset, count, hipMemcpyDeviceToHost,
					cuda::Context::getStream(context));
			assert(status == hipSuccess);

		}
	}
} /* namespace ml */

